
#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" void matmul_cuda(int n, const float *A, const float *B, float *C) {
    hipblasHandle_t h; hipblasCreate(&h);
    const float alf = 1.0f, bet = 0.0f;
    hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alf, B, n, A, n, &bet, C, n);
    hipDeviceSynchronize();
    hipblasDestroy(h);
}
